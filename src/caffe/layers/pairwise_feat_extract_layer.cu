#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/layers/pairwise_feat_extract_layer.hpp"

namespace caffe {

template <typename Dtype>
__global__ void PairwiseFeatureExtract(const int nthreads, const Dtype* const bottom,
    const int num, const int channels, const int height,
    const int width, Dtype* const top) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    // find out the local offset
    const int w = index % width;
    const int h = (index / width) % height;
    const int step = height * width;
    /* uncomment following lines and comment sequent two lines to handle n > 0 */
    /* commented for speed */
    //const int n = index / width / height;
    //int bottom_offset, top_offset;
    //if (n == 0)
    //{
    // 	bottom_offset = h*width + w;
    //	top_offset = 0;  
    //}
    //else
    //{
    //	bottom_offset = (n*channels*height + h)*width + w;  
    //	top_offset = n*(width*(4*height-3) - 3*height + 2);
    //}
    const int bottom_offset = h*width + w;
    const int top_offset = 0;
    const Dtype* const bottom_off = bottom + bottom_offset; 
    Dtype* const top_off = top + top_offset; 
    const int width_minus_1 = width - 1;
    const int height_minus_1 = height - 1;
    int c = 0;
    Dtype diff;

    if (h != 0 && w != 0 && h != height_minus_1 && w != width_minus_1)
    {
             Dtype* top_pointer = top_off + h*(4*width-3) + 4*w - 1;
	     *(top_pointer) = 0; *(top_pointer+1) = 0; *(top_pointer+2) = 0; *(top_pointer+3) = 0;
             while (c < channels)
             {
                const Dtype* const bottom_pointer = bottom_off + c*step; 
                diff = *(bottom_pointer) - *(bottom_pointer + 1);		*(top_pointer)   += diff*diff;
                diff = *(bottom_pointer) - *(bottom_pointer + width_minus_1);	*(top_pointer+1) += diff*diff;
		diff = *(bottom_pointer) - *(bottom_pointer + width);		*(top_pointer+2) += diff*diff;
		diff = *(bottom_pointer) - *(bottom_pointer + width + 1);	*(top_pointer+3) += diff*diff;
                ++c;
	     }
    }
    else if (h == 0)
    {
        if (w != 0 && w != width_minus_1)
	{
             Dtype* top_pointer = top_off + 4*w - 1;
	     *(top_pointer) = 0; *(top_pointer+1) = 0; *(top_pointer+2) = 0; *(top_pointer+3) = 0;
             while (c < channels) 
             {
                const Dtype* const bottom_pointer = bottom_off + c*step;
                diff = *(bottom_pointer) - *(bottom_pointer + 1);		*(top_pointer)   += diff*diff;
                diff = *(bottom_pointer) - *(bottom_pointer + width_minus_1);	*(top_pointer+1) += diff*diff;
		diff = *(bottom_pointer) - *(bottom_pointer + width);		*(top_pointer+2) += diff*diff;
		diff = *(bottom_pointer) - *(bottom_pointer + width + 1);	*(top_pointer+3) += diff*diff;
                ++c;
             }
        }
        else if (w == 0)
        {
	     Dtype* top_pointer = top_off;
	     *(top_pointer) = 0; *(top_pointer+1) = 0; *(top_pointer+2) = 0;
             while (c < channels)
             {
                const Dtype* const bottom_pointer = bottom_off + c*step;
                diff = *(bottom_pointer) - *(bottom_pointer + 1);		*(top_pointer)   += diff*diff;
		diff = *(bottom_pointer) - *(bottom_pointer + width);		*(top_pointer+1) += diff*diff;
		diff = *(bottom_pointer) - *(bottom_pointer + width + 1);	*(top_pointer+2) += diff*diff;
                ++c;
	     }	
        }	  	
        else
        {
             Dtype* top_pointer = top_off + 4*width - 5;
	     *(top_pointer) = 0; *(top_pointer+1) = 0;
             while (c < channels) 
             {
                const Dtype* const bottom_pointer = bottom_off + c*step; 
                diff = *(bottom_pointer) - *(bottom_pointer + width_minus_1);	*(top_pointer)   += diff*diff;
		diff = *(bottom_pointer) - *(bottom_pointer + width);		*(top_pointer+1) += diff*diff;
                ++c;	
             }
        }
    }
    else if (w == 0 && h != height_minus_1)
    {
             Dtype* top_pointer = top_off + h*(4*width-3);
	     *(top_pointer) = 0; *(top_pointer+1) = 0; *(top_pointer+2) = 0;
             while (c < channels) 
             {
                const Dtype* const bottom_pointer = bottom_off + c*step; 
                diff = *(bottom_pointer) - *(bottom_pointer + 1);		*(top_pointer)   += diff*diff;
		diff = *(bottom_pointer) - *(bottom_pointer + width);		*(top_pointer+1) += diff*diff;
		diff = *(bottom_pointer) - *(bottom_pointer + width + 1);	*(top_pointer+2) += diff*diff;
                ++c;
             }
    }
    else if (h != height_minus_1 && w == width_minus_1)
    {
             Dtype* top_pointer = top_off + 4*width*(h+1) - 3*h - 5;
	     *(top_pointer) = 0; *(top_pointer+1) = 0;
             while (c < channels) 
             {
                const Dtype* const bottom_pointer = bottom_off + c*step;
                diff = *(bottom_pointer) - *(bottom_pointer + width_minus_1);	*(top_pointer)   += diff*diff;
		diff = *(bottom_pointer) - *(bottom_pointer + width);		*(top_pointer+1) += diff*diff;
                ++c;
             }
    }
    else if (w != width_minus_1)
    {
             Dtype* top_pointer = top_off + 4*width*(height_minus_1) - 3*height + w + 3;
	     *(top_pointer) = 0;
             while (c < channels)
             {
                const Dtype* const bottom_pointer = bottom_off + c*step; 
                diff = *(bottom_pointer) - *(bottom_pointer + 1);		*(top_pointer)   += diff*diff;
                ++c;
             }
    }
  }
}

template <typename Dtype>
void PairwiseLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {

  CrossChannelForward_gpu(bottom, top);
}

template <typename Dtype>
void PairwiseLayer<Dtype>::CrossChannelForward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {

  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();

  // We will launch one kernel for each pixel location, and have the kernel
  // go through all the channels.
  int n_threads = num * height * width;

  // NOLINT_NEXT_LINE(whitespace/operators)
  PairwiseFeatureExtract<<<CAFFE_GET_BLOCKS(n_threads), CAFFE_CUDA_NUM_THREADS>>>(
      n_threads, bottom_data, num, channels, height, width, top_data);
  CUDA_POST_KERNEL_CHECK;
}

template void PairwiseLayer<float>::CrossChannelForward_gpu(
    const vector<Blob<float>*>& bottom, const vector<Blob<float>*>& top);
template void PairwiseLayer<double>::CrossChannelForward_gpu(
    const vector<Blob<double>*>& bottom, const vector<Blob<double>*>& top);

template <typename Dtype>
void PairwiseLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {

}

INSTANTIATE_LAYER_GPU_FUNCS(PairwiseLayer);

}  // namespace caffe
