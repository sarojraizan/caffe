#include "hip/hip_runtime.h"
#include <algorithm>
#include <vector>

#include "caffe/layers/trelu_layer.hpp"

namespace caffe {

template <typename Dtype>
__global__ void TReLUForward(const int n, const Dtype* bottom_data, Dtype* top_data,
    Dtype lb, Dtype ub) {
  CUDA_KERNEL_LOOP(index, n) {
    if (bottom_data[index] < lb)
	top_data[index] = lb;
    else if(bottom_data[index] > ub)
	top_data[index] = ub;
    else 
        top_data[index] = bottom_data[index];
  }
}

template <typename Dtype>
void TReLULayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  const int count = bottom[0]->count();
  Dtype ub = this->layer_param_.trelu_param().ub();
  Dtype lb = this->layer_param_.trelu_param().lb();

  // NOLINT_NEXT_LINE(whitespace/operators)
  TReLUForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, bottom_data, top_data, lb, ub);
  CUDA_POST_KERNEL_CHECK;
  // << " count: " << count << " bottom_data: "
  //     << (unsigned long)bottom_data
  //     << " top_data: " << (unsigned long)top_data
  //     << " blocks: " << CAFFE_GET_BLOCKS(count)
  //     << " threads: " << CAFFE_CUDA_NUM_THREADS;
}

template <typename Dtype>
__global__ void TReLUBackward(const int n, const Dtype* top_diff,
    const Dtype* bottom_data, Dtype* bottom_diff, Dtype lb, Dtype ub) {
  CUDA_KERNEL_LOOP(index, n) {
    if (bottom_data[index] > lb && bottom_data[index] < ub) bottom_diff[index] = top_diff[index];
    else bottom_diff[index] = Dtype(0.0);
  }
}

template <typename Dtype>
void TReLULayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {
  if (propagate_down[0]) {
    const Dtype* bottom_data = bottom[0]->gpu_data();
    const Dtype* top_diff = top[0]->gpu_diff();
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
    const int count = bottom[0]->count();
    Dtype ub = this->layer_param_.trelu_param().ub();
    Dtype lb = this->layer_param_.trelu_param().lb();

    // NOLINT_NEXT_LINE(whitespace/operators)
    TReLUBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, top_diff, bottom_data, bottom_diff, lb, ub);
    CUDA_POST_KERNEL_CHECK;
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(TReLULayer);
}  // namespace caffe
