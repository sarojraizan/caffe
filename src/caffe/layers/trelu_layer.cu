#include "hip/hip_runtime.h"
#include <algorithm>
#include <vector>

#include "caffe/layers/trelu_layer.hpp"

namespace caffe {

template <typename Dtype>
__global__ void TReLUForward(const int n, const Dtype* in, Dtype* out,
    Dtype lb, Dtype ub) {
  CUDA_KERNEL_LOOP(index, n) {
    if (in[index] < lb)
	out[index] = lb;
    else if(in[index] > ub)
	out[index] = ub;
    else 
        out[index] = in[index];
  }
}

template <typename Dtype>
void TReLULayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  const int count = bottom[0]->count();
  Dtype ub = this->layer_param_.trelu_param().ub();
  Dtype lb = this->layer_param_.trelu_param().lb();

  // NOLINT_NEXT_LINE(whitespace/operators)
  TReLUForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, bottom_data, top_data, lb, ub);
  CUDA_POST_KERNEL_CHECK;
  // << " count: " << count << " bottom_data: "
  //     << (unsigned long)bottom_data
  //     << " top_data: " << (unsigned long)top_data
  //     << " blocks: " << CAFFE_GET_BLOCKS(count)
  //     << " threads: " << CAFFE_CUDA_NUM_THREADS;
}

template <typename Dtype>
__global__ void TReLUBackward(const int n, const Dtype* in_diff,
    const Dtype* in_data, Dtype* out_diff, Dtype lb, Dtype ub) {
  CUDA_KERNEL_LOOP(index, n) {
    out_diff[index] = in_diff[index] * (in_data[index] > lb && in_data[index] < ub);
  }
}

template <typename Dtype>
void TReLULayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {
  if (propagate_down[0]) {
    const Dtype* bottom_data = bottom[0]->gpu_data();
    const Dtype* top_diff = top[0]->gpu_diff();
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
    const int count = bottom[0]->count();
    Dtype ub = this->layer_param_.trelu_param().ub();
    Dtype lb = this->layer_param_.trelu_param().lb();

    // NOLINT_NEXT_LINE(whitespace/operators)
    TReLUBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, top_diff, bottom_data, bottom_diff, lb, ub);
    CUDA_POST_KERNEL_CHECK;
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(TReLULayer);
}  // namespace caffe
