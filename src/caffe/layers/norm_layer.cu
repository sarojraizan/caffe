#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/layers/norm_layer.hpp"

namespace caffe {

template <typename Dtype>
__global__ void NormFillScale(const int nthreads, const Dtype* const in,
    const int num, const int channels, const int height,
    const int width, Dtype* const scale) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    // find out the local offset
    const int w = index % width;
    const int h = (index / width) % height;
    const int n = index / width / height;
    const int offset = (n * channels * height + h) * width + w;
    const int step = height * width;
    const Dtype* const in_off = in + offset;
    Dtype* const scale_off = scale + offset;
    int head = 0;
    Dtype accum_scale = 0;

    // square and sum the data in each channel
    while (head < channels) {
      accum_scale += in_off[head * step] * in_off[head * step];
      ++head;
    }

    head = 0;

    // add epsilon to prevent division by zero
    if (accum_scale == 0) accum_scale+=Dtype(1e-4);

    // save accumulated data in scale blob
    while (head < channels) {
      scale_off[head * step] = accum_scale;
      ++head;
    }
  }
}


template <typename Dtype>
void NormLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {

  CrossChannelForward_gpu(bottom, top);
}

// TODO: check if it would be faster to just put it into the previous kernel.
template <typename Dtype>
__global__ void NormComputeOutput(const int nthreads, const Dtype* const in,
    const Dtype* const scale, Dtype* const out, Dtype neg_beta) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    out[index] = in[index] * pow(scale[index], neg_beta);
  }
}

template <typename Dtype>
void NormLayer<Dtype>::CrossChannelForward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
  // First, compute scale
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  Dtype* scale_data = scale_.mutable_gpu_data();
  Dtype neg_beta = -0.5;
  // We will launch one kernel for each pixel location, and have the kernel
  // go through all the channels.
  int n_threads = num_ * height_ * width_;
  // NOLINT_NEXT_LINE(whitespace/operators)
  NormFillScale<<<CAFFE_GET_BLOCKS(n_threads), CAFFE_CUDA_NUM_THREADS>>>(
      n_threads, bottom_data, num_, channels_, height_, width_, scale_data);
  CUDA_POST_KERNEL_CHECK;
  n_threads = bottom[0]->count();
  // NOLINT_NEXT_LINE(whitespace/operators)
  NormComputeOutput<<<CAFFE_GET_BLOCKS(n_threads), CAFFE_CUDA_NUM_THREADS>>>(
      n_threads, bottom_data, scale_data, top_data, neg_beta);
  CUDA_POST_KERNEL_CHECK;
}

template void NormLayer<float>::CrossChannelForward_gpu(
    const vector<Blob<float>*>& bottom, const vector<Blob<float>*>& top);
template void NormLayer<double>::CrossChannelForward_gpu(
    const vector<Blob<double>*>& bottom, const vector<Blob<double>*>& top);


template <typename Dtype>
void NormLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {

  CrossChannelBackward_gpu(top, propagate_down, bottom);
}

template <typename Dtype>
__global__ void NormComputeDiff(const int nthreads,
    const Dtype* const bottom_data, const Dtype* const top_data, 
    const Dtype* const scale, const Dtype* const top_diff, const int num, const int channels, 
    const int height, const int width, Dtype* const bottom_diff, Dtype neg_beta) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    // find out the local offset
    const int w = index % width;
    const int h = (index / width) % height;
    const int n = index / width / height;
    const int offset = (n * channels * height + h) * width + w;
    const int step = height * width;
    const Dtype* const bottom_off = bottom_data + offset;
    const Dtype* const top_off = top_data + offset;
    const Dtype* const scale_off = scale + offset;
    const Dtype* const top_diff_off = top_diff + offset;
    Dtype* const bottom_diff_off = bottom_diff + offset;
    Dtype dot = Dtype(0);
    int head = 0;

    // compute bottom_diff = top_diff * scale_data^(-0.5) - dot(top_diff, top_data) / scale_data * bottom_data 
    // first compute dot = dot(top_diff, top_data)
    while (head < channels) {

      dot = dot + (top_diff_off[head * step] * top_off[head * step]);
      ++head;
    }
    head = 0;
    // now compute bottom_diff = top_diff * scale_data^(-0.5) - dot / scale_data * bottom_data
    while (head < channels) {

      bottom_diff_off[head * step] = (top_diff_off[head * step] * pow(scale_off[head * step], neg_beta)) -
                                     (bottom_off[head * step] * dot / scale_off[head * step]);
      ++head;
    }
  }
}

template <typename Dtype>
void NormLayer<Dtype>::CrossChannelBackward_gpu(
    const vector<Blob<Dtype>*>& top, const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {
  int n_threads = num_ * height_ * width_;
  Dtype neg_beta = -0.5;
  // NOLINT_NEXT_LINE(whitespace/operators)
  NormComputeDiff<<<CAFFE_GET_BLOCKS(n_threads), CAFFE_CUDA_NUM_THREADS>>>(
      n_threads, bottom[0]->gpu_data(), top[0]->gpu_data(), scale_.gpu_data(), top[0]->gpu_diff(), 
      num_, channels_, height_, width_, bottom[0]->mutable_gpu_diff(), neg_beta);
}
template void NormLayer<float>::CrossChannelBackward_gpu(
    const vector<Blob<float>*>& top, const vector<bool>& propagate_down,
    const vector<Blob<float>*>& bottom);
template void NormLayer<double>::CrossChannelBackward_gpu(
    const vector<Blob<double>*>& top, const vector<bool>& propagate_down,
    const vector<Blob<double>*>& bottom);

INSTANTIATE_LAYER_GPU_FUNCS(NormLayer);
}  // namespace caffe
