#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/layers/pairwise_avg_layer.hpp"

namespace caffe {

template <typename Dtype>
__global__ void PairwiseAvgFeatureExtract(const int nthreads, const Dtype* const bottom,
    const int num, const int channels, const int height,
    const int width, Dtype* const top) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    // find out the local offset
    const int w = index % width;
    const int h = (index / width) % height;
    /* uncomment following lines and comment sequent two lines to handle n > 0 */
    /* commented for speed */
    //const int n = index / width / height;
    //int bottom_offset, top_offset;
    //if (n == 0)
    //{
    // 	bottom_offset = h*width + w;
    //	top_offset = 0;  
    //}
    //else
    //{
    //	bottom_offset = (n*channels*height + h)*width + w;  
    //	top_offset = n*(width*(4*height-3) - 3*height + 2);
    //}
    const int bottom_offset = h*width + w;
    const int top_offset = 0;
    const Dtype* const bottom_off = bottom + bottom_offset; 
    Dtype* const top_off = top + top_offset; 
    const int width_minus_1 = width - 1;
    const int height_minus_1 = height - 1;
    Dtype sum;

    if (h != 0 && w != 0 && h != height_minus_1 && w != width_minus_1)
    {
             	Dtype* top_pointer = top_off + h*(4*width-3) + 4*w - 1;
                const Dtype* const bottom_pointer = bottom_off; 
                sum = *(bottom_pointer) + *(bottom_pointer + 1);		*(top_pointer)   = sum*Dtype(0.5);
                sum = *(bottom_pointer) + *(bottom_pointer + width_minus_1);	*(top_pointer+1) = sum*Dtype(0.5);
		sum = *(bottom_pointer) + *(bottom_pointer + width);		*(top_pointer+2) = sum*Dtype(0.5);
		sum = *(bottom_pointer) + *(bottom_pointer + width + 1);	*(top_pointer+3) = sum*Dtype(0.5);
    }
    else if (h == 0)
    {
        if (w != 0 && w != width_minus_1)
	{
             	Dtype* top_pointer = top_off + 4*w - 1;
                const Dtype* const bottom_pointer = bottom_off;
                sum = *(bottom_pointer) + *(bottom_pointer + 1);		*(top_pointer)   = sum*Dtype(0.5);
                sum = *(bottom_pointer) + *(bottom_pointer + width_minus_1);	*(top_pointer+1) = sum*Dtype(0.5);
		sum = *(bottom_pointer) + *(bottom_pointer + width);		*(top_pointer+2) = sum*Dtype(0.5);
		sum = *(bottom_pointer) + *(bottom_pointer + width + 1);	*(top_pointer+3) = sum*Dtype(0.5);
        }
        else if (w == 0)
        {
	     	Dtype* top_pointer = top_off;
                const Dtype* const bottom_pointer = bottom_off;
                sum = *(bottom_pointer) + *(bottom_pointer + 1);		*(top_pointer)   = sum*Dtype(0.5);
		sum = *(bottom_pointer) + *(bottom_pointer + width);		*(top_pointer+1) = sum*Dtype(0.5);
		sum = *(bottom_pointer) + *(bottom_pointer + width + 1);	*(top_pointer+2) = sum*Dtype(0.5);
        }	  	
        else
        {
             	Dtype* top_pointer = top_off + 4*width - 5;
                const Dtype* const bottom_pointer = bottom_off; 
                sum = *(bottom_pointer) + *(bottom_pointer + width_minus_1);	*(top_pointer)   = sum*Dtype(0.5);
		sum = *(bottom_pointer) + *(bottom_pointer + width);		*(top_pointer+1) = sum*Dtype(0.5);
        }
    }
    else if (w == 0 && h != height_minus_1)
    {
             	Dtype* top_pointer = top_off + h*(4*width-3);
                const Dtype* const bottom_pointer = bottom_off; 
                sum = *(bottom_pointer) + *(bottom_pointer + 1);		*(top_pointer)   = sum*Dtype(0.5);
		sum = *(bottom_pointer) + *(bottom_pointer + width);		*(top_pointer+1) = sum*Dtype(0.5);
		sum = *(bottom_pointer) + *(bottom_pointer + width + 1);	*(top_pointer+2) = sum*Dtype(0.5);
    }
    else if (h != height_minus_1 && w == width_minus_1)
    {
             	Dtype* top_pointer = top_off + 4*width*(h+1) - 3*h - 5;
                const Dtype* const bottom_pointer = bottom_off;
                sum = *(bottom_pointer) + *(bottom_pointer + width_minus_1);	*(top_pointer)   = sum*Dtype(0.5);
		sum = *(bottom_pointer) + *(bottom_pointer + width);		*(top_pointer+1) = sum*Dtype(0.5);

    }
    else if (w != width_minus_1)
    {
             	Dtype* top_pointer = top_off + 4*width*(height_minus_1) - 3*height + w + 3;
                const Dtype* const bottom_pointer = bottom_off; 
                sum = *(bottom_pointer) + *(bottom_pointer + 1);		*(top_pointer)   = sum*Dtype(0.5);
    }
  }
}

template <typename Dtype>
void PairwiseAvgLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {

  CrossChannelForward_gpu(bottom, top);
}

template <typename Dtype>
void PairwiseAvgLayer<Dtype>::CrossChannelForward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {

  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();

  // We will launch one kernel for each pixel location, and have the kernel
  // go through all the channels.
  int n_threads = num * height * width;

  // NOLINT_NEXT_LINE(whitespace/operators)
  PairwiseAvgFeatureExtract<<<CAFFE_GET_BLOCKS(n_threads), CAFFE_CUDA_NUM_THREADS>>>(
      n_threads, bottom_data, num, channels, height, width, top_data);
  CUDA_POST_KERNEL_CHECK;
}

template void PairwiseAvgLayer<float>::CrossChannelForward_gpu(
    const vector<Blob<float>*>& bottom, const vector<Blob<float>*>& top);
template void PairwiseAvgLayer<double>::CrossChannelForward_gpu(
    const vector<Blob<double>*>& bottom, const vector<Blob<double>*>& top);

template <typename Dtype>
void PairwiseAvgLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {

}

INSTANTIATE_LAYER_GPU_FUNCS(PairwiseAvgLayer);

}  // namespace caffe
