#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/util/io.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/layers/dot_product_loss_layer.hpp"

namespace caffe {

template <typename Dtype>
__global__ void SetTempForMissingGT(const int nthreads, const Dtype* const bottom,
    const Dtype* const label, const int num, const int channels, 
    const int height, const int width, Dtype* const temp) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    // find out the local offset
    const int w = index % width;
    const int h = (index / width) % height;
    const int n = index / width / height;
    const int offset = (n * channels * height + h) * width + w;
    const int offset_temp = (n * height + h) * width + w;
    const int step = height * width;
    const Dtype* const label_off = label + offset;
    const Dtype* const bottom_off = bottom + offset;
    Dtype* const temp_off = temp + offset_temp;
    int head = 0;
    Dtype mask(0);
    
    // calculate dot product if groundtruth is present, else set dot product to 1
    // the sum of all the (three) channels in the bottom labels blob == 0
    // implies that the groundtruth data is missing
    while (head < channels) {
      mask = label_off[head * step] + mask;
      ++head;
    }
    if (mask != Dtype(0)){
        temp_off[0] = bottom_off[0] * label_off[0];
        head = 1;
	while (head < channels) {
           temp_off[0] += bottom_off[head * step] * label_off[head * step];
	   ++head;
	}
    }
    else
    {
        temp_off[0] = 1;
    }
  }
}

template <typename Dtype>
void DotProductLossLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  Dtype* temp = temp_.mutable_gpu_data();
  const Dtype* ones = ones_.gpu_data();
  const Dtype* bottom_data = bottom[0]->gpu_data();
  const Dtype* label = bottom[1]->gpu_data();

  // calculate dot product if groundtruth is present, 
  // else set dot product to 1 (parallelized on gpu)
  // Note: the bottom[1] blob should contain the groundtruth labels
  int n_threads = num_ * height_ * width_;
  // NOLINT_NEXT_LINE(whitespace/operators)
  SetTempForMissingGT<<<CAFFE_GET_BLOCKS(n_threads), CAFFE_CUDA_NUM_THREADS>>>(
      n_threads, bottom_data, label, num_, channels_, height_, width_, temp);

  Dtype dot;
  caffe_gpu_dot(ones_.count(), ones, temp, &dot);
  Dtype loss = (height_ * width_) - dot / num_;
  top[0]->mutable_cpu_data()[0] = loss;
}

template <typename Dtype>
void DotProductLossLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  if (propagate_down[0]) {
    const Dtype sign(-1);
    const Dtype alpha = sign * top[0]->cpu_diff()[0] / num_;
    caffe_gpu_axpby(
        bottom[0]->count(),              // count
        alpha,                           // a
        bottom[1]->gpu_data(),           // x
        Dtype(0),                        // b
        bottom[0]->mutable_gpu_diff());  // y
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(DotProductLossLayer);
}  // namespace caffe
