#include "hip/hip_runtime.h"
#include <vector>
#include <sstream>
#include "caffe/layer.hpp"
#include "caffe/util/io.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/layers/sparse_depth_euclidean_loss_layer.hpp"

namespace caffe {

template <typename Dtype>
__global__ void SetDiffZeroForMissingGTDepth(const int nthreads,
    const Dtype* const label, const int num, 
    const int height, const int width, Dtype* const diff) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    // find out the local offset
    const int w = index % width;
    const int h = (index / width) % height;
    const int n = index / width / height;
    const int offset = (n * height + h) * width + w;
    const Dtype* const label_off = label + offset;
    Dtype* const diff_off = diff + offset;
    
    // set diff_ = 0 if groundtruth data is missing
    // the channel in the bottom labels blob == 0.0
    // implies that the groundtruth data is missing
    Dtype mask = label_off[0];

    if (mask == Dtype(0.0))
        diff_off[0] = Dtype(0.0);
  }
}

template <typename Dtype>
__global__ void SetBottomDiffZeroForMissingGTDepth(const int nthreads,
    const Dtype* const label, const int num, 
    const int height, const int width, Dtype* const bottom_diff) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    // find out the local offset
    const int w = index % width;
    const int h = (index / width) % height;
    const int n = index / width / height;
    const int offset = (n * height + h) * width + w;
    const Dtype* const label_off = label + offset;
    Dtype* const bottom_diff_off = bottom_diff + offset;
    
    // set diff_ = 0 if groundtruth data is missing
    // the channel in the bottom labels blob == 0.0
    // implies that the groundtruth data is missing
    Dtype mask = label_off[0];

    if (mask == Dtype(0.0))
        bottom_diff_off[0] = Dtype(0.0);
  }
}

template <typename Dtype>
__global__ void ComputeLogDepths(const int nthreads,
    const Dtype* const label, const int num, 
    const int height, const int width, Dtype* const logdepths) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    // find out the local offset
    const int w = index % width;
    const int h = (index / width) % height;
    const int n = index / width / height;
    const int offset = (n * height + h) * width + w;
    const Dtype* const label_off = label + offset;
    Dtype* const logdepths_off = logdepths + offset;
    
    Dtype mask = label_off[0];

    if (mask != Dtype(0.0))
        logdepths_off[0] = log(mask)/Dtype(0.45723134);
    else
        logdepths_off[0] = Dtype(0.0);
  }
}

template <typename Dtype>
__global__ void ComputeDDiff(const int nthreads,
    const Dtype* const diff, const int num, 
    const int height, const int width, Dtype* const ddiff_x, Dtype* const ddiff_y) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    // find out the local offset
    const int w = index % width;
    const int h = (index / width) % height;
    const int n = index / width / height;
    const int offset = (n * height + h) * width + w;
    const Dtype* const diff_off = diff + offset;
    Dtype* const ddiff_x_off = ddiff_x + offset;
    Dtype* const ddiff_y_off = ddiff_y + offset;
    
    if (w != (width - 1)) *(ddiff_x_off) = *(diff_off + 1) - *(diff_off); 
    else *(ddiff_x_off) = Dtype(0.0);
    if (h < (height - 1)) *(ddiff_y_off) = *(diff_off + width) - *(diff_off);
    else *(ddiff_y_off) = Dtype(0.0);
  }
}

template <typename Dtype>
__global__ void ComputeDiv(const int nthreads,
    const Dtype N, const int height, const int width, const Dtype* const ddiff_x, const Dtype* const ddiff_y, Dtype* const bottom_diff, Dtype top_diff_val) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    // find out the local offset
    const int w = index % width;
    const int h = (index / width) % height;
    const int n = index / width / height;
    const int offset = (n * height + h) * width + w;
    const Dtype* const ddiff_x_off = ddiff_x + offset;
    const Dtype* const ddiff_y_off = ddiff_y + offset;
    Dtype* const bottom_diff_off = bottom_diff + offset;
    
    if (w != 0) *(bottom_diff_off) += top_diff_val*Dtype(2.0)/N*(*(ddiff_x_off-1) - *(ddiff_x_off));
    else *(bottom_diff_off) += top_diff_val*Dtype(-2.0)/N*(*ddiff_x_off);
    if (h != 0) *(bottom_diff_off) += top_diff_val*Dtype(2.0)/N*(*(ddiff_y_off - width) - *(ddiff_y_off));
    else *(bottom_diff_off) += top_diff_val*Dtype(-2.0)/N*(*ddiff_y_off);
  }
}

template <typename Dtype>
void SparseDepthEuclideanLossLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  int count = bottom[0]->count();
  int num = bottom[0]->num();
  int height = bottom[0]->height();
  int width = bottom[0]->width();

  int n_threads = num * height * width;
  ComputeLogDepths<<<CAFFE_GET_BLOCKS(n_threads), CAFFE_CUDA_NUM_THREADS>>>(
      n_threads, bottom[1]->gpu_data(), num, height, width, logdepths_.mutable_gpu_data());

  caffe_gpu_sub(
      count,
      bottom[0]->gpu_data(),
      logdepths_.gpu_data(),
      diff_.mutable_gpu_data());

  // set diff_ = 0 if groundtruth data is missing (parallelized on gpu)
  // Note: the bottom[1] blob should contain the groundtruth labels

  // NOLINT_NEXT_LINE(whitespace/operators)
  SetDiffZeroForMissingGTDepth<<<CAFFE_GET_BLOCKS(n_threads), CAFFE_CUDA_NUM_THREADS>>>(
      n_threads, bottom[1]->gpu_data(), num, height, width, diff_.mutable_gpu_data());

  // NOLINT_NEXT_LINE(whitespace/operators)
  ComputeDDiff<<<CAFFE_GET_BLOCKS(n_threads), CAFFE_CUDA_NUM_THREADS>>>(
      n_threads, diff_.gpu_data(), num, height, width, ddiff_x_.mutable_gpu_data(), ddiff_y_.mutable_gpu_data());

  Dtype dot, ddiff_x2_sum, ddiff_y2_sum;
  caffe_gpu_dot(count, diff_.gpu_data(), diff_.gpu_data(), &dot);
  caffe_gpu_dot(count, diff_.gpu_data(), ones_.gpu_data(), &diff_sum);
  caffe_gpu_dot(count, ddiff_x_.gpu_data(), ddiff_x_.gpu_data(), &ddiff_x2_sum);
  caffe_gpu_dot(count, ddiff_y_.gpu_data(), ddiff_y_.gpu_data(), &ddiff_y2_sum);

  Dtype N = bottom[0]->num();
  Dtype loss = dot / N          
               + ddiff_x2_sum / N 
               + ddiff_y2_sum / N
               - diff_sum*diff_sum / (Dtype(2.0)*N*N)
               ;
  top[0]->mutable_cpu_data()[0] = loss;
}

template <typename Dtype>
void SparseDepthEuclideanLossLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
   
 if (propagate_down[0]) {

     int count = bottom[0]->count();
     int num = bottom[0]->num();
     int height = bottom[0]->height();
     int width = bottom[0]->width();

     int n_threads = num * height * width;
     Dtype N = bottom[0]->num();
     Dtype top_diff_val = top[0]->cpu_diff()[0];

     caffe_gpu_axpby(
         count,              
         top_diff_val*Dtype(2.0)/N,       // a
         diff_.gpu_data(),                // x
         Dtype(0.0),                      // b
         bottom[0]->mutable_gpu_diff());  // y

     caffe_gpu_axpby(
         count,              
         top_diff_val*Dtype(-1.0)/(N*N)*diff_sum,      // a
         ones_.gpu_data(),                             // x
         Dtype(1.0),                                   // b
         bottom[0]->mutable_gpu_diff());               // y
     
     ComputeDiv<<<CAFFE_GET_BLOCKS(n_threads), CAFFE_CUDA_NUM_THREADS>>>(
         n_threads, N, height, width, ddiff_x_.gpu_data(), ddiff_y_.gpu_data(), bottom[0]->mutable_gpu_diff(), top_diff_val);
     SetBottomDiffZeroForMissingGTDepth<<<CAFFE_GET_BLOCKS(n_threads), CAFFE_CUDA_NUM_THREADS>>>(
         n_threads, bottom[1]->gpu_data(), num, height, width, bottom[0]->mutable_gpu_diff());	
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(SparseDepthEuclideanLossLayer);

}  // namespace caffe
