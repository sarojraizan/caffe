#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/util/io.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/layers/sparse_depth_euclidean_loss_layer.hpp"

namespace caffe {

template <typename Dtype>
__global__ void SetDiffZeroForMissingGTDepth(const int nthreads,
    const Dtype* const label, const int num, 
    const int height, const int width, Dtype* const diff) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    // find out the local offset
    const int w = index % width;
    const int h = (index / width) % height;
    const int n = index / width / height;
    const int offset = (n * height + h) * width + w;
    const Dtype* const label_off = label + offset;
    Dtype* const diff_off = diff + offset;
    
    // set diff_ = 0 if groundtruth data is missing
    // the channel in the bottom labels blob == -10.0
    // implies that the groundtruth data is missing
    Dtype mask = label_off[0];

    if (mask == Dtype(-10.0))
        diff_off[0] = Dtype(0);
  }
}

template <typename Dtype>
void SparseDepthEuclideanLossLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  int count = bottom[0]->count();
  int num = bottom[0]->num();
  int height = bottom[0]->height();
  int width = bottom[0]->width();

  caffe_gpu_sub(
      count,
      bottom[0]->gpu_data(),
      bottom[1]->gpu_data(),
      diff_.mutable_gpu_data());

  // set diff_ = 0 if groundtruth data is missing (parallelized on gpu)
  // Note: the bottom[1] blob should contain the groundtruth labels
  int n_threads = num * height * width;
  // NOLINT_NEXT_LINE(whitespace/operators)
  SetDiffZeroForMissingGTDepth<<<CAFFE_GET_BLOCKS(n_threads), CAFFE_CUDA_NUM_THREADS>>>(
      n_threads, bottom[1]->gpu_data(), num, height, width, diff_.mutable_gpu_data());

  Dtype dot;
  caffe_gpu_dot(count, diff_.gpu_data(), diff_.gpu_data(), &dot);
  Dtype loss = dot / num / Dtype(2);
  top[0]->mutable_cpu_data()[0] = loss;
}

template <typename Dtype>
void SparseDepthEuclideanLossLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {

 if (propagate_down[0]) {
     const Dtype alpha = top[0]->cpu_diff()[0] / bottom[0]->num();
     caffe_gpu_axpby(
         bottom[0]->count(),              // count
         alpha,                           // a
         diff_.gpu_data(),                // x
         Dtype(0),                        // b
         bottom[0]->mutable_gpu_diff());  // y
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(SparseDepthEuclideanLossLayer);

}  // namespace caffe
