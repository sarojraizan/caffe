#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/util/io.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/layers/sparse_euclidean_loss_layer.hpp"

namespace caffe {

template <typename Dtype>
__global__ void SetDiffZeroForMissingGT(const int nthreads,
    const Dtype* const label, const int num, const int channels, 
    const int height, const int width, Dtype* const diff) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    // find out the local offset
    const int w = index % width;
    const int h = (index / width) % height;
    const int n = index / width / height;
    const int offset = (n * channels * height + h) * width + w;
    const int step = height * width;
    const Dtype* const label_off = label + offset;
    Dtype* const diff_off = diff + offset;
    int head = 0;
    Dtype mask(0);
    
    // set diff_ = 0 if groundtruth data is missing
    // the sum of all the (three) channels in the bottom labels blob == 0
    // implies that the groundtruth data is missing
    while (head < channels) {
      mask = label_off[head * step] + mask;
      ++head;
    }
    if (mask == Dtype(0)){
        head = 0;
	while (head < channels) {
           diff_off[head * step] = Dtype(0);
	   ++head;
	}
    }
  }
}

template <typename Dtype>
void SparseEuclideanLossLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  int count = bottom[0]->count();
  int num = bottom[0]->num();
  int channels = bottom[0]->channels();
  int height = bottom[0]->height();
  int width = bottom[0]->width();

  caffe_gpu_sub(
      count,
      bottom[0]->gpu_data(),
      bottom[1]->gpu_data(),
      diff_.mutable_gpu_data());

  // set diff_ = 0 if groundtruth data is missing (parallelized on gpu)
  // Note: the bottom[1] blob should contain the groundtruth labels
  int n_threads = num * height * width;
  // NOLINT_NEXT_LINE(whitespace/operators)
  SetDiffZeroForMissingGT<<<CAFFE_GET_BLOCKS(n_threads), CAFFE_CUDA_NUM_THREADS>>>(
      n_threads, bottom[1]->gpu_data(), num, channels, height, width, diff_.mutable_gpu_data());

  Dtype dot;
  caffe_gpu_dot(count, diff_.gpu_data(), diff_.gpu_data(), &dot);
  Dtype loss = dot / num / Dtype(2);
  top[0]->mutable_cpu_data()[0] = loss;
}

template <typename Dtype>
void SparseEuclideanLossLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {

 if (propagate_down[0]) {
     const Dtype alpha = top[0]->cpu_diff()[0] / bottom[0]->num();
     caffe_gpu_axpby(
         bottom[0]->count(),              // count
         alpha,                           // a
         diff_.gpu_data(),                // x
         Dtype(0),                        // b
         bottom[0]->mutable_gpu_diff());  // y
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(SparseEuclideanLossLayer);

}  // namespace caffe
